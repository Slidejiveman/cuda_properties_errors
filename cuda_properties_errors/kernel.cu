
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

int main()
{
	// Fetch device properties and display them to the screen.
	int nDevices;

	// All CUDA API calls have a return value that indicate
	// whether or not an error occurred during the execution
	// of the function.
	hipError_t err = hipGetDeviceCount(&nDevices);
	// Code like this will handle errors in the CPU calls.
	// Kernel errors are more difficult to handle than this
	// since they are executing asynchronously with respect
	// to the host (CPU). Debugging macros in the in the 
	// asynchronous portions of your code prevents concurrency,
	// so be wise when checking those. Probably not good for 
	// release builds.
	if (err != hipSuccess) {
		printf("%s\n", hipGetErrorString(err));
	}

	for (int i = 0; i < nDevices; i++) {
		hipDeviceProp_t prop; // Note: this struct has many other fields
		hipGetDeviceProperties(&prop, i);
		printf("Device Number: %d\n", i);
		printf("  Device name: %s\n", prop.name);
		printf("  Memory Clock Rate (KHz): %d\n", prop.memoryClockRate);
		printf("  Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
		
		// This calculation represents the theoretical peak memory bandwidth
		// which is based on the given hardware specs.
		printf("  Peak Memory Bandwidth (GB/s): %f\n\n", 
			2.0 * prop.memoryClockRate * (prop.memoryBusWidth / 8) / 1.0e6);
	}
    return 0;
}
